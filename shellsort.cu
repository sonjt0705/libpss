#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include "shellsort.h"

#define THREAD_COUNT 1024
#define THREAD_DUMMY 1023

__global__ void gap_shell_sort(int* data, UINT32 n, UINT32 g) {
	UINT32 i = blockIdx.x * blockDim.x + threadIdx.x;
	int t;
	UINT32 k;
	if (i >= g) return;
	for (UINT32 j = i; j < n; j += g) {
		t = data[j];
		k = j;
		while (k > g - 1 && data[k - g] > t) {
			data[k] = data[k - g];
			k -= g;
		}
		data[k] = t;
	}
}

void shell_sort(int* data, UINT32 n, gap_scale s, clock_t* e) {
	UINT32 bd = THREAD_COUNT;
	UINT32 gd = (n + THREAD_DUMMY) / THREAD_COUNT;
	clock_t sg = clock();
	for (UINT32 g = n / s; g > 1; g /= s) gap_shell_sort <<< gd, bd >>> (data, n, g);
	clock_t fg = clock();
	hipDeviceSynchronize();
	int t;
	UINT32 k;
	clock_t so = clock();
	for (UINT32 j = 0; j < n; j++) {
		t = data[j];
		k = j;
		while (k > 0 && data[k - 1] > t) {
			data[k] = data[k - 1];
			k--;
		}
		data[k] = t;
	}
	clock_t fo = clock();
	if (e != nullptr) *e = (fg - sg) + (fo - so);
}
