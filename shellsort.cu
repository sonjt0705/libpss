#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include "shellsort.h"
#include <chrono>

#define THREAD_COUNT 1024
#define THREAD_DUMMY 1023

using namespace std::chrono;

__global__ void gap_shell_sort(int* data, UINT32 n, UINT32 g) {
	UINT32 i = blockIdx.x * blockDim.x + threadIdx.x;
	int t;
	UINT32 k;
	if (i >= g) return;
	for (UINT32 j = i; j < n; j += g) {
		t = data[j];
		k = j;
		while (k > g - 1 && data[k - g] > t) {
			data[k] = data[k - g];
			k -= g;
		}
		data[k] = t;
	}
}

void shell_sort(int* data, UINT32 n, gap_scale s, INT64* e) {
	UINT32 bd = THREAD_COUNT;
	UINT32 gd = (n + THREAD_DUMMY) / THREAD_COUNT;
	system_clock::time_point sg = system_clock::now();
	for (UINT32 g = n / s; g > 1; g /= s) gap_shell_sort <<< gd, bd >>> (data, n, g);
	system_clock::time_point fg = system_clock::now();
	hipDeviceSynchronize();
	int t;
	UINT32 k;
	system_clock::time_point so = system_clock::now();
	for (UINT32 j = 0; j < n; j++) {
		t = data[j];
		k = j;
		while (k > 0 && data[k - 1] > t) {
			data[k] = data[k - 1];
			k--;
		}
		data[k] = t;
	}
	system_clock::time_point fo = system_clock::now();
	if (e != nullptr) *e = duration_cast<microseconds>(fg - sg).count() + duration_cast<microseconds>(fo - so).count();
}
